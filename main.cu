#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <chrono>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <string>

#define CUDACHECK(cmd)                                         \
    do {                                                       \
        hipError_t e = cmd;                                   \
        if (e != hipSuccess) {                                \
            printf("Failed: Cuda error %s:%d '%s'\n",          \
                   __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

#define NCCLCHECK(cmd)                                         \
    do {                                                       \
        ncclResult_t r = cmd;                                  \
        if (r != ncclSuccess) {                                \
            printf("Failed, NCCL error %s:%d '%s'\n",          \
                   __FILE__, __LINE__, ncclGetErrorString(r)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

__global__ void kernel(float *buf, int sz) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= sz)
        return;
    for (int j = 0; j < sz; ++j) {
        buf[i] += buf[j];
    }
}

int main(int argc, char *argv[]) {
    ncclComm_t comms[2];

    //managing 2 devices
    int nDev = 2;
    size_t comp_size = 50 * 1024;
    int nccl_size = 10 * 1024 * 1024;
    int devs[2] = {0, 1};
    int num_threads = 1024;

    //allocating and initializing device buffers
    auto nccl_buff = new float *[nDev];
    auto comp_buff = new float *[nDev];
    auto nccl_streams = new hipStream_t[nDev];
    auto comp_streams = new hipStream_t[nDev];

    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipMalloc(&nccl_buff[i], nccl_size * sizeof(float)));
        CUDACHECK(hipMalloc(&comp_buff[i], comp_size * sizeof(float)));
        CUDACHECK(hipStreamCreate(&nccl_streams[i]));
        CUDACHECK(hipStreamCreate(&comp_streams[i]));
    }

    //initializing NCCL
    NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

    hipProfilerStart();
    printf("start timing\n");
    auto start_ts = std::chrono::high_resolution_clock::now();

    //calling NCCL communication API. Group API is required when using
    //multiple devices per thread
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < nDev; ++i) {
        NCCLCHECK(ncclAllReduce(nccl_buff[i], nccl_buff[i], nccl_size, ncclFloat, ncclSum, comms[i], nccl_streams[i], 2, 288));
    }

    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        kernel<<<comp_size / num_threads, num_threads, 0, comp_streams[i]>>>(comp_buff[i], comp_size);
        CUDACHECK(hipPeekAtLastError());
    }
    NCCLCHECK(ncclGroupEnd());

    //synchronizing on CUDA streams to wait for completion of NCCL operation
    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipStreamSynchronize(nccl_streams[i]));
        CUDACHECK(hipStreamSynchronize(comp_streams[i]));
    }

    auto end_ts = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_ts - start_ts).count();
    printf("Time taken: %ld milliseconds\n", duration);

    hipProfilerStop();

    //finalizing NCCL
    for (int i = 0; i < nDev; ++i) {
        ncclCommDestroy(comms[i]);
    }

    //free device buffers
    for (int i = 0; i < nDev; ++i) {
        CUDACHECK(hipSetDevice(i));
        CUDACHECK(hipFree(nccl_buff[i]));
        CUDACHECK(hipFree(comp_buff[i]));
    }
}